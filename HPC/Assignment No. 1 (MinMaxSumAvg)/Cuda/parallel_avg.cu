
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>

#define SIZE 100

__global__ void sum(const int* __restrict__ input, const int size, int* sumOut)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    atomicAdd(sumOut, input[i]);
    __syncthreads();
}

int main()
{
  int i;
  int a[SIZE];
  int c = 0;
  int *dev_a, *dev_c;
  float gpu_elapsed_time,avg;
  hipEvent_t gpu_start,gpu_stop;
    
  hipMalloc((void **) &dev_a, SIZE*sizeof(int));
  hipMalloc((void **) &dev_c, sizeof(int));
  srand(time(0));
  for( i = 0 ; i < SIZE ; i++)
  {
    a[i] = (rand() % (1000 - 100 + 1)) + 100;
  }
  for( i = 0 ; i < SIZE ; i++)
  {
    printf("%d ",a[i]);
    if (i%10==0 && i!=0){
      printf("\n");
    }
  }
  hipMemcpy(dev_c , &c, sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dev_a , a, SIZE*sizeof(int),hipMemcpyHostToDevice);
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_stop);
  hipEventRecord(gpu_start,0);
  sum<<<2,SIZE/2>>>(dev_a,SIZE,dev_c);
  hipDeviceSynchronize();
  hipMemcpy(&c, dev_c, sizeof(int),hipMemcpyDeviceToHost);
  c = c / SIZE;
  hipEventRecord(gpu_stop, 0);
  hipEventSynchronize(gpu_stop);
  hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
  hipEventDestroy(gpu_start);
  hipEventDestroy(gpu_stop);
  printf("avg =  %d ",c);
  printf("\nThe gpu took: %f milli-seconds.\n",gpu_elapsed_time);
    
  printf("\n");
  printf("avg =  %d ",c);
  hipFree(dev_a);
  hipFree(dev_c);
  return 0;
}
