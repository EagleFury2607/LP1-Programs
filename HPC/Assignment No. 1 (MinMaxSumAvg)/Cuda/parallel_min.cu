
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define SIZE 100

__global__ void min(const int* __restrict__ input, const int size, int* minOut)
{
    int localMin = 1000;

    for (int i = threadIdx.x; i < size; i += blockDim.x)
    {
        int val = input[i];
        if (localMin > abs(val))
        {
            localMin = abs(val);
        }
    }
    atomicMin(minOut, localMin);
    __syncthreads();
}

int main()
{
  int i;
  int a[SIZE];
  int c;
  int *dev_a, *dev_c;
  hipMalloc((void **) &dev_a, SIZE*sizeof(int));
  hipMalloc((void **) &dev_c, sizeof(int));
  srand(time(0));
  for( i = 0 ; i < SIZE ; i++)
  {
    a[i] = (rand() % (1000 - 100 + 1)) + 100;
  }
  for( i = 0 ; i < SIZE ; i++)
  {
    printf("%d ",a[i]);
    if (i%10==0 && i!=0){
      printf("\n");
    }
  }
  c = a[0];
  hipMemcpy(dev_c , &c, sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dev_a , a, SIZE*sizeof(int),hipMemcpyHostToDevice);
  min<<<2,SIZE/2>>>(dev_a,SIZE,dev_c);
  hipDeviceSynchronize();
  hipMemcpy(&c, dev_c, sizeof(int),hipMemcpyDeviceToHost);
  printf("\n");
  printf("min =  %d ",c);
  hipFree(dev_a);
  hipFree(dev_c);
  return 0;
}
