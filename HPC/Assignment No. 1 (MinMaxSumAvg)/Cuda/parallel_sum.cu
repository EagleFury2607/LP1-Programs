
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>

#define SIZE 100

__global__ void sum(const int* __restrict__ input, const int size, int* sumOut)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    atomicAdd(sumOut, input[i]);
    __syncthreads();
}

int main()
{
  int i;
  int a[SIZE];
  int c = 0;
  int *dev_a, *dev_c;
  hipMalloc((void **) &dev_a, SIZE*sizeof(int));
  hipMalloc((void **) &dev_c, sizeof(int));
  srand(time(0));
  for( i = 0 ; i < SIZE ; i++)
  {
    a[i] = (rand() % (1000 - 100 + 1)) + 100;
  }
  for( i = 0 ; i < SIZE ; i++)
  {
    printf("%d ",a[i]);
    if (i%10==0 && i!=0){
      printf("\n");
    }
  }
  hipMemcpy(dev_c , &c, sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dev_a , a, SIZE*sizeof(int),hipMemcpyHostToDevice);
  sum<<<2,SIZE/2>>>(dev_a,SIZE,dev_c);
  hipDeviceSynchronize();
  hipMemcpy(&c, dev_c, sizeof(int),hipMemcpyDeviceToHost);
  printf("\n");
  printf("sum =  %d ",c);
  hipFree(dev_a);
  hipFree(dev_c);
  return 0;
}
